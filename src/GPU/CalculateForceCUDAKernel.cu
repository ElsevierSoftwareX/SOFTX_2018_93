
#include <hip/hip_runtime.h>
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"

using namespace cub;

void CallBoxInterForceGPU(VariablesCUDA *vars,
			  vector<uint> pair1,
			  vector<uint> pair2,
			  XYZArray const &currentCoords,
			  XYZArray const &currentCOM,
			  BoxDimensions const& boxAxes,
			  bool electrostatic,
			  vector<double> particleCharge,
			  vector<int> particleKind,
			  vector<int> particleMol,
			  double &rT11,
			  double &rT12,
			  double &rT13,
			  double &rT22,
			  double &rT23,
			  double &rT33,
			  double &vT11,
			  double &vT12,
			  double &vT13,
			  double &vT22,
			  double &vT23,
			  double &vT33,
			  uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int *gpu_pair1, *gpu_pair2;
  int *gpu_particleKind;
  int *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_x, *gpu_y, *gpu_z;
  double *gpu_comx, *gpu_comy, *gpu_comz;
  double *gpu_rT11, *gpu_rT12, *gpu_rT13;
  double *gpu_rT22, *gpu_rT23, *gpu_rT33;
  double *gpu_vT11, *gpu_vT12, *gpu_vT13;
  double *gpu_vT22, *gpu_vT23, *gpu_vT33;
  double *gpu_final_value;

  cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int));
  cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int));
  cudaMalloc((void**) &gpu_x, atomNumber * sizeof(double));
  cudaMalloc((void**) &gpu_y, atomNumber * sizeof(double));
  cudaMalloc((void**) &gpu_z, atomNumber * sizeof(double));
  cudaMalloc((void**) &gpu_particleCharge, 
	     particleCharge.size() * sizeof(double));
  cudaMalloc((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  cudaMalloc((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  cudaMalloc((void**) &gpu_rT11, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_rT12, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_rT13, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_rT22, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_rT23, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_rT33, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT11, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT12, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT13, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT22, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT23, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_vT33, pair1.size() * sizeof(double));
  cudaMalloc((void**) &gpu_final_value, sizeof(double));
  cudaMalloc((void**) &gpu_comx, molNumber * sizeof(double));
  cudaMalloc((void**) &gpu_comy, molNumber * sizeof(double));
  cudaMalloc((void**) &gpu_comz, molNumber * sizeof(double));

  cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_x, currentCoords.x, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_y, currentCoords.y, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_z, currentCoords.z, atomNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_comx, currentCOM.x, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_comy, currentCOM.y, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_comz, currentCOM.z, molNumber * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0],
	     particleCharge.size() * sizeof(double),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0],
	     particleKind.size() * sizeof(int),
	     cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0],
	     particleMol.size() * sizeof(int),
	     cudaMemcpyHostToDevice);

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size()/threadsPerBlock) + 1;
  BoxInterForceGPU<<<blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
						       gpu_pair2,
						       gpu_x,
						       gpu_y,
						       gpu_z,
						       gpu_comx,
						       gpu_comy,
						       gpu_comz,
						       boxAxes.GetAxis(box).x,
						       boxAxes.GetAxis(box).y,
						       boxAxes.GetAxis(box).z,
						       electrostatic,
						       gpu_particleCharge,
						       gpu_particleKind,
						       gpu_particleMol,
						       gpu_rT11,
						       gpu_rT12,
						       gpu_rT13,
						       gpu_rT22,
						       gpu_rT23,
						       gpu_rT33,
						       gpu_vT11,
						       gpu_vT12,
						       gpu_vT13,
						       gpu_vT22,
						       gpu_vT23,
						       gpu_vT33,
						       pair1.size(),
						       vars->gpu_sigmaSq,
						       vars->gpu_epsilon_Cn,
						       vars->gpu_n,
						       vars->gpu_VDW_Kind,
						       vars->gpu_isMartini,
						       vars->gpu_count,
						       vars->gpu_rCut,
						       vars->gpu_rCutLow,
						       vars->gpu_rOn,
						       vars->gpu_alpha,
						       vars->gpu_ewald,
						       vars->gpu_diElectric_1);

  cudaError_t err;
  err = cudaGetLastError();
  if(err != cudaSuccess)
  {
    std::cout << cudaGetErrorString(err) << std::endl;
    exit(-1);
  }
  // ReduceSum // Virial of LJ
  void * d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT11,
		    gpu_final_value, pair1.size());
  cudaMalloc(&d_temp_storage, temp_storage_bytes);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT11,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT11, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT12,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT12, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT13,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT13, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT22,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT22, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT23,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT23, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_vT33,
		    gpu_final_value, pair1.size());
  cudaMemcpy(&vT33, gpu_final_value, sizeof(double),
	     cudaMemcpyDeviceToHost);

  if(electrostatic)
  {
    // ReduceSum // Virial of Coulomb
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT11,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT11, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT12,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT12, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT13,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT13, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT22,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT22, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT23,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT23, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
    DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_rT33,
		      gpu_final_value, pair1.size());
    cudaMemcpy(&rT33, gpu_final_value, sizeof(double),
	       cudaMemcpyDeviceToHost);
  }

  cudaFree(d_temp_storage);
  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_x);
  cudaFree(gpu_y);
  cudaFree(gpu_z);
  cudaFree(gpu_comx);
  cudaFree(gpu_comy);
  cudaFree(gpu_comz);
  cudaFree(gpu_final_value);
  cudaFree(gpu_rT11);
  cudaFree(gpu_rT12);
  cudaFree(gpu_rT13);
  cudaFree(gpu_rT22);
  cudaFree(gpu_rT23);
  cudaFree(gpu_rT33);
  cudaFree(gpu_vT11);
  cudaFree(gpu_vT12);
  cudaFree(gpu_vT13);
  cudaFree(gpu_vT22);
  cudaFree(gpu_vT23);
  cudaFree(gpu_vT33);
}

__global__ void BoxInterForceGPU(int *gpu_pair1,
				 int *gpu_pair2,
				 double *gpu_x,
				 double *gpu_y,
				 double *gpu_z,
				 double *gpu_comx,
				 double *gpu_comy,
				 double *gpu_comz,
				 double xAxes,
				 double yAxes,
				 double zAxes,
				 bool electrostatic,
				 double *gpu_particleCharge,
				 int *gpu_particleKind,
				 int *gpu_particleMol,
				 double *gpu_rT11,
				 double *gpu_rT12,
				 double *gpu_rT13,
				 double *gpu_rT22,
				 double *gpu_rT23,
				 double *gpu_rT33,
				 double *gpu_vT11,
				 double *gpu_vT12,
				 double *gpu_vT13,
				 double *gpu_vT22,
				 double *gpu_vT23,
				 double *gpu_vT33,
				 int pairSize,
				 double *gpu_sigmaSq,
				 double *gpu_epsilon_Cn,
				 double *gpu_n,
				 int *gpu_VDW_Kind,
				 int *gpu_isMartini,
				 int *gpu_count,
				 double *gpu_rCut,
				 double *gpu_rCutLow,
				 double *gpu_rOn,
				 double *gpu_alpha,
				 int *gpu_ewald,
				 double *gpu_diElectric_1)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID >= pairSize)
    return;
  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  //tensors for VDW and real part of electrostatic
  gpu_vT11[threadID] = 0.0, gpu_vT22[threadID] = 0.0, gpu_vT33[threadID] = 0.0;
  gpu_rT11[threadID] = 0.0, gpu_rT22[threadID] = 0.0, gpu_rT33[threadID] = 0.0;
  // extra tensors reserved for later on
  gpu_vT12[threadID] = 0.0, gpu_vT13[threadID] = 0.0, gpu_vT23[threadID] = 0.0;
  gpu_rT12[threadID] = 0.0, gpu_rT13[threadID] = 0.0, gpu_rT23[threadID] = 0.0;
  double diff_comx, diff_comy, diff_comz;

  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]], 
	       gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]], 
	       gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]], 
	       gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes/2.0, 
	       yAxes/2.0, zAxes/2.0, gpu_rCut[0]))
  {
    diff_comx = gpu_comx[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comx[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comy = gpu_comy[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comy[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comz = gpu_comz[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comz[gpu_particleMol[gpu_pair2[threadID]]];

    diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes/2.0);
    diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes/2.0);
    diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes/2.0);

    if(electrostatic)
    {
      qi_qj = gpu_particleCharge[gpu_pair1[threadID]] * 
	gpu_particleCharge[gpu_pair2[threadID]];
      pRF = CalcCoulombForceGPU(distSq, qi_qj, gpu_VDW_Kind[0], gpu_ewald[0],
				gpu_isMartini[0], gpu_alpha[0], gpu_rCut[0],
				gpu_diElectric_1[0]);
      
      gpu_rT11[threadID] = pRF * (virX * diff_comx);
      gpu_rT22[threadID] = pRF * (virY * diff_comy);
      gpu_rT33[threadID] = pRF * (virZ * diff_comz);
      
      //extra tensor calculations
      gpu_rT12[threadID] = pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
      gpu_rT13[threadID] = pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
      gpu_rT23[threadID] = pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
    }
    
    pVF = CalcEnForceGPU(distSq, gpu_particleKind[gpu_pair1[threadID]], 
			 gpu_particleKind[gpu_pair2[threadID]],
			 gpu_sigmaSq, gpu_n, gpu_epsilon_Cn, gpu_rCut[0],
			 gpu_rOn[0], gpu_isMartini[0], gpu_VDW_Kind[0],
			 gpu_count[0]);
    
    gpu_vT11[threadID] = pVF * (virX * diff_comx);
    gpu_vT22[threadID] = pVF * (virY * diff_comy);
    gpu_vT33[threadID] = pVF * (virZ * diff_comz);
      
    //extra tensor calculations
    gpu_vT12[threadID] = pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
    gpu_vT13[threadID] = pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
    gpu_vT23[threadID] = pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
  }
}

__device__ double CalcCoulombForceGPU(double distSq, double qi_qj,
				      int gpu_VDW_Kind, int gpu_ewald,
				      int gpu_isMartini, double gpu_alpha,
				      double gpu_rCut, double gpu_diElectric_1)
{
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcCoulombVirParticleGPU(distSq, qi_qj, gpu_alpha);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcCoulombVirShiftGPU(distSq, qi_qj, gpu_ewald, gpu_alpha);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcCoulombVirSwitchMartiniGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
					  gpu_rCut, gpu_diElectric_1);
  }
  else
    return CalcCoulombVirSwitchGPU(distSq, qi_qj, gpu_ewald, gpu_alpha,
				   gpu_rCut);
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2,
				 double *gpu_sigmaSq, double *gpu_n,
				 double *gpu_epsilon_Cn, double gpu_rCut,
				 double gpu_rOn, int gpu_isMartini,
				 int gpu_VDW_Kind, int gpu_count)
{
  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcVirParticleGPU(distSq, index, gpu_sigmaSq, gpu_n,
			      gpu_epsilon_Cn);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcVirShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcVirSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
				   gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  }
  else
    return CalcVirSwitchGPU(distSq, index, gpu_sigmaSq, gpu_epsilon_Cn, gpu_n,
			    gpu_rCut, gpu_rOn);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj,
					    double gpu_alpha)
{
  double dist = sqrt(distSq);
  double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
  double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
  double temp = 1.0 - erf(gpu_alpha * dist);
  return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj,
					 int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double dist = sqrt(distSq);
    return qi_qj/(distSq * dist);
  }
}
__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq, double qi_qj,
						 int gpu_ewald,
						 double gpu_alpha,
						 double gpu_rCut,
						 double gpu_diElectric_1)
{
  if(gpu_ewald)
  {
     double dist = sqrt(distSq);
     double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
     double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
     double temp = 1.0 - erf(gpu_alpha * dist);
     return  qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
     // in Martini, the Coulomb switching distance is zero, so we will have
     // sqrt(distSq) - rOnCoul =  sqrt(distSq)
     double dist = sqrt(distSq);
     double rij_ronCoul_2 = distSq;
     double rij_ronCoul_3 = dist * distSq;
     
     double A1 = 1.0 * (-(1.0+4)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					   pow(gpu_rCut, 2));
     double B1 = -1.0 * (-(1.0+3)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					    pow(gpu_rCut, 3));

     double virCoul = A1/rij_ronCoul_2 + B1/rij_ronCoul_3;
     return qi_qj * gpu_diElectric_1 * ( 1.0/(dist * distSq) + virCoul/dist);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj,
					  int gpu_ewald, double gpu_alpha,
					  double gpu_rCut)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq/rCutSq - 1.0;
    switchVal *= switchVal;

    double dSwitchVal = 2.0 * (distSq/rCutSq - 1.0) * 2.0 * dist/rCutSq;
    return -1.0 * qi_qj * (dSwitchVal/distSq - switchVal/(distSq * dist));
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index,
				     double *gpu_sigmaSq, double *gpu_n,
				     double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirShiftGPU(double distSq, int index, double *gpu_sigmaSq,
				  double *gpu_n, double *gpu_epsilon_Cn)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index,
					  double *gpu_sigmaSq, double *gpu_n,
					  double *gpu_epsilon_Cn,
					  double gpu_rCut, double gpu_rOn)
{
  double r_1 = 1.0/sqrt(distSq);
  double r_8 = pow(r_1, 8);
  double r_n2 = pow(r_1, gpu_n[index]+2);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;

  double pn = gpu_n[index];
  double An = pn * ((pn+1)*gpu_rOn - (pn+4)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 2));
  double Bn = -pn * ((pn+1)*gpu_rOn-(pn+3)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 3));

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn/2);

  double A6 = 6.0 * ((6.0+1)*gpu_rOn-(6.0+4)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 2));
  double B6 = -6.0 * ((6.0+1)*gpu_rOn-(6.0+3)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 3));

  double dshifttempRep = An * rij_ron_2 + Bn * rij_ron_3;
  double dshifttempAtt = A6 * rij_ron_2 + B6 * rij_ron_3;
  
  const double dshiftRep = ( distSq > gpu_rOn * gpu_rOn ? 
			     dshifttempRep * r_1 : 0);
  const double dshiftAtt = ( distSq > gpu_rOn * gpu_rOn ?
			     dshifttempAtt * r_1 : 0);
  double Wij = gpu_epsilon_Cn[index] * (sign * (pn * r_n2 + dshiftRep) -
					sig6 * (6.0 * r_8 + dshiftAtt));
  return Wij;
}

__device__ double CalcVirSwitchGPU(double distSq, int index,
				   double *gpu_sigmaSq, double *gpu_epsilon_Cn,
				   double *gpu_n, double gpu_rCut,
				   double gpu_rOn)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rCutSq_rijSq = rCutSq - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rNeg2 = 1.0/distSq;
  double rRat2 = rNeg2 * gpu_sigmaSq[index];
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);

  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);
  double fW = 12.0 * factor2 * rCutSq_rijSq * (rOnSq - distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);
  const double factW = ( distSq > rOnSq ? fW : 0.0);

  double Wij = gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
  double Eij = gpu_epsilon_Cn[index] * (repulse - attract);

  return (Wij * factE - Eij * factW);
}

#endif
