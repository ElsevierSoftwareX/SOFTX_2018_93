
#include <hip/hip_runtime.h>
#ifdef GOMC_CUDA

#include <cuda.h>
#include "CalculateForceCUDAKernel.cuh"
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "cub/cub.cuh"

using namespace cub;

void CallBoxInterForceGPU(vector<uint> pair1,
			  vector<uint> pair2,
			  XYZArray const &currentCoords,
			  XYZArray const &currentCOM,
			  BoxDimensions const& boxAxes,
			  bool electrostatic,
			  vector<double> particleCharge,
			  vector<int> particleKind,
			  vector<int> particleMol,
			  double &virInter,
			  double &virReal,
			  uint const box)
{
  int atomNumber = currentCoords.Count();
  int molNumber = currentCOM.Count();
  int *gpu_pair1, *gpu_pair2;
  int *gpu_particleKind;
  int *gpu_particleMol;
  int blocksPerGrid, threadsPerBlock;
  double *gpu_particleCharge;
  double *gpu_x, *gpu_y, *gpu_z;
  double *gpu_comx, *gpu_comy, *gpu_comz;
  double *gpu_virInter, *gpu_virReal;
  double *gpu_final_virInter, *gpu_final_virReal;

  CubDebugExit(cudaMalloc((void**) &gpu_pair1, pair1.size() * sizeof(int)));
  CubDebugExit(cudaMalloc((void**) &gpu_pair2, pair2.size() * sizeof(int)));
  CubDebugExit(cudaMalloc((void**) &gpu_x, atomNumber * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_y, atomNumber * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_z, atomNumber * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_particleCharge, 
			  particleCharge.size() * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_particleKind, 
			  particleKind.size() * sizeof(int)));
  CubDebugExit(cudaMalloc((void**) &gpu_particleMol, 
			  particleMol.size() * sizeof(int)));
  CubDebugExit(cudaMalloc((void**) &gpu_virInter, 
			  pair1.size() * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_virReal, 
			  pair1.size() * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_final_virReal, 
			  pair1.size() * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_final_virInter, 
			  pair1.size() * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_comx, molNumber * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_comy, molNumber * sizeof(double)));
  CubDebugExit(cudaMalloc((void**) &gpu_comz, molNumber * sizeof(double)));

  CubDebugExit(cudaMemcpy(gpu_pair1, &pair1[0], pair1.size() * sizeof(int),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_pair2, &pair2[0], pair2.size() * sizeof(int),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_x, currentCoords.x, atomNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_y, currentCoords.y, atomNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_z, currentCoords.z, atomNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_comx, currentCOM.x, molNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_comy, currentCOM.y, molNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_comz, currentCOM.z, molNumber * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_particleCharge, &particleCharge[0],
			  particleCharge.size() * sizeof(double),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_particleKind, &particleKind[0],
			  particleKind.size() * sizeof(int),
			  cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_particleMol, &particleMol[0],
			  particleMol.size() * sizeof(int),
			  cudaMemcpyHostToDevice));

  // Run the kernel...
  threadsPerBlock = 256;
  blocksPerGrid = (int)(pair1.size()/threadsPerBlock) + 1;
  BoxInterForceGPU<<<blocksPerGrid, threadsPerBlock>>>(gpu_pair1,
						       gpu_pair2,
						       gpu_x,
						       gpu_y,
						       gpu_z,
						       gpu_comx,
						       gpu_comy,
						       gpu_comz,
						       boxAxes.GetAxis(box).x,
						       boxAxes.GetAxis(box).y,
						       boxAxes.GetAxis(box).z,
						       electrostatic,
						       gpu_particleCharge,
						       gpu_particleKind,
						       gpu_particleMol,
						       gpu_virInter,
						       gpu_virReal,
						       pair1.size());

  // ReduceSum // Virial of LJ
  void * d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_virInter,
		    gpu_final_virInter, pair1.size());
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_virInter,
		    gpu_final_virInter, pair1.size());
  cudaFree(d_temp_storage);

  // ReduceSum // Virial of Coulomb
  d_temp_storage = NULL;
  temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_virReal,
		    gpu_final_virReal, pair1.size());
  CubDebugExit(cudaMalloc(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_virReal,
		    gpu_final_virReal, pair1.size());
  cudaFree(d_temp_storage);
  
  // Copy back the result to CPU ! :)
  CubDebugExit(cudaMemcpy(&virInter, gpu_final_virInter, sizeof(double),
			  cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(&virReal, gpu_final_virReal, sizeof(double),
			  cudaMemcpyDeviceToHost));
  
  cudaFree(gpu_pair1);
  cudaFree(gpu_pair2);
  cudaFree(gpu_particleKind);
  cudaFree(gpu_particleMol);
  cudaFree(gpu_particleCharge);
  cudaFree(gpu_x);
  cudaFree(gpu_y);
  cudaFree(gpu_z);
  cudaFree(gpu_comx);
  cudaFree(gpu_comy);
  cudaFree(gpu_comz);
  cudaFree(gpu_virReal);
  cudaFree(gpu_virInter);
  cudaFree(gpu_final_virReal);
  cudaFree(gpu_final_virInter);
}

__global__ void BoxInterForceGPU(int *gpu_pair1,
				 int *gpu_pair2,
				 double *gpu_x,
				 double *gpu_y,
				 double *gpu_z,
				 double *gpu_comx,
				 double *gpu_comy,
				 double *gpu_comz,
				 double xAxes,
				 double yAxes,
				 double zAxes,
				 bool electrostatic,
				 double *gpu_particleCharge,
				 int *gpu_particleKind,
				 int *gpu_particleMol,
				 double *gpu_virInter,
				 double *gpu_virReal,
				 int pairSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID > pairSize)
    return;
  double distSq;
  double virX, virY, virZ;
  double pRF = 0.0, qi_qj, pVF = 0.0;
  //tensors for VDW and real part of electrostatic
  double vT11 = 0.0, vT12 = 0.0, vT13 = 0.0;
  double vT22 = 0.0, vT23 = 0.0, vT33 = 0.0;
  double rT11 = 0.0, rT12 = 0.0, rT13 = 0.0;
  double rT22 = 0.0, rT23 = 0.0, rT33 = 0.0;
  double diff_comx, diff_comy, diff_comz;

  if(InRcutGPU(distSq, virX, virY, virZ, gpu_x[gpu_pair1[threadID]], 
	       gpu_y[gpu_pair1[threadID]], gpu_z[gpu_pair1[threadID]], 
	       gpu_x[gpu_pair2[threadID]], gpu_y[gpu_pair2[threadID]], 
	       gpu_z[gpu_pair2[threadID]], xAxes, yAxes, zAxes, xAxes/2.0, 
	       yAxes/2.0, zAxes/2.0))
  {
    diff_comx = gpu_comx[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comx[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comy = gpu_comy[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comy[gpu_particleMol[gpu_pair2[threadID]]];
    diff_comz = gpu_comz[gpu_particleMol[gpu_pair1[threadID]]] - 
      gpu_comz[gpu_particleMol[gpu_pair2[threadID]]];

    diff_comx = MinImageSignedGPU(diff_comx, xAxes, xAxes/2.0);
    diff_comy = MinImageSignedGPU(diff_comy, yAxes, yAxes/2.0);
    diff_comz = MinImageSignedGPU(diff_comz, zAxes, zAxes/2.0);

    if(electrostatic)
    {
      qi_qj = gpu_particleCharge[gpu_pair1[threadID]] * 
	gpu_particleCharge[gpu_pair2[threadID]];
      pRF = CalcCoulombForceGPU(distSq, qi_qj);
      
      rT11 = pRF * (virX * diff_comx);
      rT22 = pRF * (virY * diff_comy);
      rT33 = pRF * (virZ * diff_comz);
      
      //extra tensor calculations
      rT12 = pRF * (0.5 * (virX * diff_comy + virY * diff_comx));
      rT13 = pRF * (0.5 * (virX * diff_comz + virZ * diff_comx));
      rT23 = pRF * (0.5 * (virY * diff_comz + virZ * diff_comy));
    }
    
    pVF = CalcEnForceGPU(distSq, gpu_particleKind[gpu_pair1[threadID]], 
			 gpu_particleKind[gpu_pair2[threadID]]);
    
    vT11 = pVF * (virX * diff_comx);
    vT22 = pVF * (virY * diff_comy);
    vT33 = pVF * (virZ * diff_comz);
      
    //extra tensor calculations
    vT12 = pVF * (0.5 * (virX * diff_comy + virY * diff_comx));
    vT13 = pVF * (0.5 * (virX * diff_comz + virZ * diff_comx));
    vT23 = pVF * (0.5 * (virY * diff_comz + virZ * diff_comy));
  }

  gpu_virInter[threadID] = vT11 + vT22 + vT33;
  gpu_virReal[threadID] = rT11 + rT22 + rT33;
}

__device__ double CalcCoulombForceGPU(double distSq, double qi_qj)
{
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcCoulombVirParticleGPU(distSq, qi_qj);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcCoulombVirShiftGPU(distSq, qi_qj);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcCoulombVirSwitchMartiniGPU(distSq, qi_qj);
  }
  else
    return CalcCoulombVirSwitchGPU(distSq, qi_qj);
}

__device__ double CalcEnForceGPU(double distSq, int kind1, int kind2)
{
  int index = FlatIndexGPU(kind1, kind2);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND)
  {
    return CalcVirParticleGPU(distSq, index);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND)
  {
    return CalcVirShiftGPU(distSq, index);
  }
  else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini)
  {
    return CalcVirSwitchMartiniGPU(distSq, index);
  }
  else
    return CalcVirSwitchGPU(distSq, index);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombVirParticleGPU(double distSq, double qi_qj)
{
  double dist = sqrt(distSq);
  double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
  double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
  double temp = 1.0 - erf(gpu_alpha * dist);
  return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
}

__device__ double CalcCoulombVirShiftGPU(double distSq, double qi_qj)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double dist = sqrt(distSq);
    return qi_qj/(distSq * dist);
  }
}
__device__ double CalcCoulombVirSwitchMartiniGPU(double distSq, double qi_qj)
{
  if(gpu_ewald)
  {
     double dist = sqrt(distSq);
     double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
     double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
     double temp = 1.0 - erf(gpu_alpha * dist);
     return  qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
     // in Martini, the Coulomb switching distance is zero, so we will have
     // sqrt(distSq) - rOnCoul =  sqrt(distSq)
     double dist = sqrt(distSq);
     double rij_ronCoul_2 = distSq;
     double rij_ronCoul_3 = dist * distSq;
     
     double A1 = 1.0 * (-(1.0+4)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					   pow(gpu_rCut, 2));
     double B1 = -1.0 * (-(1.0+3)*gpu_rCut)/(pow(gpu_rCut,1.0+2) *
					    pow(gpu_rCut, 3));

     double virCoul = A1/rij_ronCoul_2 + B1/rij_ronCoul_3;
     return qi_qj * gpu_diElectric_1 * ( 1.0/(dist * distSq) + virCoul/dist);
  }
}

__device__ double CalcCoulombVirSwitchGPU(double distSq, double qi_qj)
{
  if(gpu_ewald)
  {
    double dist = sqrt(distSq);
    double constValue = 2.0 * gpu_alpha / sqrt(M_PI);
    double expConstValue = exp(-1.0 * gpu_alpha * gpu_alpha * distSq);
    double temp = 1.0 - erf(gpu_alpha * dist);
    return qi_qj * (temp / dist + constValue * expConstValue) / distSq;
  }
  else
  {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq/rCutSq - 1.0;
    switchVal *= switchVal;

    double dSwitchVal = 2.0 * (distSq/rCutSq - 1.0) * 2.0 * dist/rCutSq;
    return -1.0 * qi_qj * (dSwitchVal/distSq - switchVal/(distSq * dist));
  }
}

//VDW Calculation
//*****************************************************************//
__device__ double CalcVirParticleGPU(double distSq, int index)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirShiftGPU(double distSq, int index)
{
  double rNeg2 = 1.0/distSq;
  double rRat2 = gpu_sigmaSq[index] * rNeg2;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  return gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
}

__device__ double CalcVirSwitchMartiniGPU(double distSq, int index)
{
  double r_1 = 1.0/sqrt(distSq);
  double r_8 = pow(r_1, 8);
  double r_n2 = pow(r_1, gpu_n[index]+2);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;

  double pn = gpu_n[index];
  double An = pn * ((pn+1)*gpu_rOn - (pn+4)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 2));
  double Bn = -pn * ((pn+1)*gpu_rOn-(pn+3)*gpu_rCut)/
    (pow(gpu_rCut, pn+2)*pow(gpu_rCut-gpu_rOn, 3));

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn/2);

  double A6 = 6.0 * ((6.0+1)*gpu_rOn-(6.0+4)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 2));
  double B6 = -6.0 * ((6.0+1)*gpu_rOn-(6.0+3)*gpu_rCut)/
    (pow(gpu_rCut,6.0+2)*pow(gpu_rCut-gpu_rOn, 3));

  double dshifttempRep = An * rij_ron_2 + Bn * rij_ron_3;
  double dshifttempAtt = A6 * rij_ron_2 + B6 * rij_ron_3;
  
  const double dshiftRep = ( distSq > gpu_rOn * gpu_rOn ? 
			     dshifttempRep * r_1 : 0);
  const double dshiftAtt = ( distSq > gpu_rOn * gpu_rOn ?
			     dshifttempAtt * r_1 : 0);
  double Wij = gpu_epsilon_Cn[index] * (sign * (pn * r_n2 + dshiftRep) -
					sig6 * (6.0 * r_8 + dshiftAtt));
  return Wij;
}

__device__ double CalcVirSwitchGPU(double distSq, int index)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rCutSq_rijSq = rCutSq - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rNeg2 = 1.0/distSq;
  double rRat2 = rNeg2 * gpu_sigmaSq[index];
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index]/2.0);
  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);

  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);
  double fW = 12.0 * factor2 * rCutSq_rijSq * (rOnSq - distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);
  const double factW = ( distSq > rOnSq ? fW : 0.0);

  double Wij = gpu_epsilon_Cn[index] * 6.0 * 
    ((gpu_n[index]/6.0) * repulse - attract) * rNeg2;
  double Eij = gpu_epsilon_Cn[index] * (repulse - attract);

  return (Wij * factE - Eij * factW);
}

#endif
