#include "hip/hip_runtime.h"
#include "CalculateEwaldCUDAKernel.h"

#ifdef GOMC_CUDA

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ConstantDefinitionsCUDA.h"
#include "CalculateMinImageCUDA.h"

void CallBoxReciprocalSetupGPU(XYZArray const & coords,
			       double const *kx,
			       double const *ky,
			       double const *kz,
			       vector<double> particleCharge,
			       uint imageSize,
			       double *sumRnew,
			       double *sumInew)
{
  double *gpu_x, *gpu_y, *gpu_z;
  double *gpu_kx, *gpu_ky, *gpu_kz;
  double *gpu_particleCharge;
  double *gpu_sumRnew, *gpu_sumInew;
  double *gpu_imageSize;
  int start, length;
  int i = 0;
  int blocksPerGrid, threadsPerBlock;
  int atomNumber = coords.Count();

  hipMalloc((void**) &gpu_x, atomNumber * sizeof(double));
  hipMalloc((void**) &gpu_y, atomNumber * sizeof(double));
  hipMalloc((void**) &gpu_z, atomNumber * sizeof(double));
  hipMalloc((void**) &gpu_particleCharge, 
	     particleCharge.size * sizeof(double));
  hipMalloc((void**) &gpu_sumRnew, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_sumInew, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_kx, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_ky, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_kz, imageSize * sizeof(double));

  hipMemcpy(gpu_particleCharge, &particleCharge[0], 
	     particleCharge.size() * sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(gpu_x, coords.x, atomNumber * sizeof(double),
	     hipMemcpyHostToDevice);
  hipMemcpy(gpu_y, coords.y, atomNumber * sizeof(double),
	     hipMemcpyHostToDevice);
  hipMemcpy(gpu_z, coords.z, atomNumber * sizeof(double),
	     hipMemcpyHostToDevice);
  hipMemcpy(gpu_kx, kx, imageSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_ky, ky, imageSize * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(gpu_kz, kz, imageSize * sizeof(double), hipMemcpyHostToDevice);

  threadsPerBlock = 256;
  blocksPerGrid = (int)(imageSize/threadsPerBlock) + 1;

  hipFree(gpu_x);
  hipFree(gpu_y);
  hipFree(gpu_z);
  hipFree(gpu_particleCharge);
  hipFree(gpu_sumRnew);
  hipFree(gpu_sumInew);
  hipFree(gpu_kx);
  hipFree(gpu_ky);
  hipFree(gpu_kz);
}

void CallBoxReciprocalGPU(double * prefact,
			  double * sumRnew,
			  double * sumInew,
			  int imageSize)
{
  double * gpu_sumRnew, *gpu_sumInew;
  double * gpu_prefact;
  double * gpu_energyRecip;
  int blockPerGrid, threadsPerBlock;
  
  hipMalloc((void**) &gpu_sumRnew, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_sumInew, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_prefact, imageSize * sizeof(double));
  hipMalloc((void**) &gpu_energyRecip, imageSize * sizeof(double));

  hipMemcpy(gpu_prefact, prefact, imageSize * sizeof(double),
	     hipMemcpyHostToDevice);
  hipMemcpy(gpu_sumRnew, sumRnew, imageSize * sizeof(double),
	     hipMemcpyHostToDevice);
  hipMemcpy(gpu_sumInew, sumInew, imageSize * sizeof(double),
	     hipMemcpyHostToDevice);
  
  threadsPerBlock = 256;
  blocksPerGrid = (imageSize/threadsPerBlock) + 1;
  BoxReciprocalGPU<<<blocksPerGrid, threadsPerBlock>>>(gpu_prefact,
						       gpu_sumRnew,
						       gpu_sumInew,
						       gpu_energyRecip,
						       imageSize);

  hipFree(gpu_sumRnew);
  hipFree(gpu_sumInew);
  hipFree(gpu_prefact);
}

__global__ void BoxReciprocalSetupGPU(double * gpu_x,
				      double * gpu_y,
				      dobule * gpu_z,
				      double * gpu_kx,
				      double * gpu_ky,
				      double * gpu_kz,
				      double atomNumber,
				      double * gpu_particleCharge,
				      double * gpu_sumRnew,
				      double * gpu_sumInew,
				      double imageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID>=imageSize)
    return;
  int i;
  double dotP;
  
  sumRnew[threadiD] = 0.0;
  sumInew[threadID] = 0.0;
  for(i=0; i<atomNumber; i++)
  {
    dotP = DotProduct(gpu_kx[threadID], gpu_ky[threadID], gpu_kz[threadID],
		      gpu_x[threadID], gpu_y[threadID], gpu_z[threadID]);
    gpu_sumRnew[threadID] += gpu_particleCharge[threadID] * cos(dotP);
    gpu_sumInew[threadID] += gpu_particleCharge[threadID] * sin(dotP);
  }

  // reduction
}

__global__ void BoxReciprocalGPU(double *gpu_prefact,
				 double *gpu_sumRnew,
				 double *gpu_sumInew,
				 double *gpu_energyRecip,
				 int mageSize)
{
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  if(threadID>=imageSize)
    return;
  
  gpu_energyRecip[threadID] = gpu_sumRnew[threadID] * gpu_sumRnew[threadID] +
    gpu_sumInew[threadID] * gpu_sumInew[threadID] * gpu_prefact[threadID];
}

__device__ double DotProduct(double kx, double ky, double kx, 
			     double x, double y, double z)
{
  return (kx * x + ky * y + kz * z);
}

#endif